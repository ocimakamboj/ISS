
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <cstdlib>
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <list>
#include <utility> 
#include <math.h>
#include <limits>
#include <ctime>
#define N 20000
using namespace std;


__global__ void reduce(int *g_idata, int* g_odata)
{
	static __shared__ int sdata[1024];

	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i =   blockIdx.x*blockDim.x + threadIdx.x;

	if(tid<544){
		for(int j=0; j<20 ; j++){
			sdata[tid] = sdata[tid] + g_idata[i+1024*j];
		}
	}else{
		for(int j=0; j<19 ; j++){
			sdata[tid] = sdata[tid] + g_idata[i+1024*j];
		}
	}
	
	__syncthreads();

	// do reduction in shared mem
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		if ((tid % (2*s)) == 0){
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0){
		g_odata[blockIdx.x] = sdata[tid];
	}
}

int* load_data1(){
	int* input_data = new int[N];

	for(int i=0;i<N;i++){
		input_data[i] = i;
		
	}
	return(input_data);
}

int* load_data(const char* input_file){
	int* answer = new int[N];
	FILE* fp = fopen(input_file,"r"); 
	int element;

	int i = 0;
	while( fscanf(fp, "%d", &element) != EOF )
    {
        answer[i] = element;
        i++;
    }
    fclose(fp);
    return(answer);
}

int main(){
	hipEvent_t start, stop;
 	float time;
 	hipEventCreate(&start);
 	hipEventCreate(&stop);

	int* h_idata;
	int* h_odata; /* host data*/
	int* d_idata;
	int* d_odata; /* device data*/

	int numThreadsperBlock = 1024;
	int numBlocks =  1;
	int size1 = N * sizeof( int );
	int size2 = numBlocks * sizeof(int);

	hipMalloc( (void **) &d_idata, size1 );
	hipMalloc( (void **) &d_odata, size2);
	h_idata = (int *)malloc( size1 );
	h_odata = (int *)malloc( size2);

	const char* name = "reduce_data.txt";

	h_idata = load_data(name);
	
	/* copying inputs to device memory */
	hipMemcpy(d_idata, h_idata, size1, hipMemcpyHostToDevice) ;
	hipMemcpy(d_odata, h_odata, size2, hipMemcpyHostToDevice) ;

	dim3 dimBlock(numThreadsperBlock, 1, 1); dim3 dimGrid(numBlocks, 1, 1);

	hipEventRecord(start, 0);
	reduce<<< dimGrid, dimBlock >>>(d_idata, d_odata);
	hipEventRecord(stop, 0);
 	hipEventSynchronize(stop);

	int result;
	hipMemcpy( h_odata, d_odata, size2, hipMemcpyDeviceToHost );
	hipMemcpy( &result, d_odata, size2, hipMemcpyDeviceToHost );
	cout << "sum is " << result << endl;

	hipEventElapsedTime(&time, start, stop);
 	printf ("Time for the kernel: %f ms\n", time);

	free(h_idata);
	free(h_odata);
	hipFree(d_idata);
	hipFree(d_odata);
}